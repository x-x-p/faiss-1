#include "hip/hip_runtime.h"
//
// Created by root on 5/7/20.
//

#include <faiss/gpu/GpuIndexInt8Flat.h>
#include <faiss/IndexInt8Flat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/I8FlatIndex.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/StaticUtils.h>
#include <limits>

namespace faiss { namespace gpu {


GpuIndexInt8Flat::GpuIndexInt8Flat(GpuResources *resources, const faiss::IndexInt8Flat *index,
                                   GpuIndexInt8FlatConfig config) :
        GpuIndexInt8(resources,
                     index->d,
                     index->metric_type,
                     config),
        config_(std::move(config)),
        data_(nullptr) {
    // Flat index doesn't need training
    this->is_trained = true;
    copyFrom(index);
}

GpuIndexInt8Flat::GpuIndexInt8Flat(GpuResources *resources, int dims,
                                   GpuIndexInt8FlatConfig config) :
        GpuIndexInt8(resources, dims, METRIC_INNER_PRODUCT, config),
        config_(std::move(config)),
        data_(nullptr) {
    // Flat index doesn't need training
    this->is_trained = true;

    // Construct index
    DeviceScope scope(device_);
    data_ = new I8FlatIndex(resources,
                            dims,
                            config_.storeTransposed,
                            memorySpace_);
}

GpuIndexInt8Flat::~GpuIndexInt8Flat() {
    delete data_;
}

void
GpuIndexInt8Flat::copyFrom(const faiss::IndexInt8Flat *index)
{
    // todo impl
}

void GpuIndexInt8Flat::copyTo(faiss::IndexInt8Flat *index) const {
    // todo impl
}

size_t
GpuIndexInt8Flat::getNumVecs() const {
    return this->ntotal;
}

void
GpuIndexInt8Flat::reset() {
    DeviceScope scope(device_);

    // Free the underlying memory
    data_->reset();
    this->ntotal = 0;
}

void
GpuIndexInt8Flat::train(Index::idx_t n, const int8_t* x) {
    // nothing to do
}

size_t
GpuIndexInt8Flat::remove_id(faiss::Index::idx_t i)
{
    if(i > ntotal - 1) {
        return 0;
    }
    DeviceScope scope(device_);
    data_->del(i, resources_->getDefaultStream(device_));
    return 1;
}

void
GpuIndexInt8Flat::update(idx_t key, const int8_t *recons)
{
    if(key > ntotal - 1) {
        return;
    }
    DeviceScope scope(device_);
    auto stream = resources_->getDefaultStream(device_);

    auto vec = data_->getVectorsRef()[key];
    toDevice(vec.data(), recons, d, stream);
}

void
GpuIndexInt8Flat::add(Index::idx_t n, const int8_t* x) {
    FAISS_THROW_IF_NOT_MSG(this->is_trained, "Index not trained");

    // For now, only support <= max int results
    FAISS_THROW_IF_NOT_FMT(n <= (Index::idx_t) std::numeric_limits<int>::max(),
                           "GPU index only supports up to %d indices",
                           std::numeric_limits<int>::max());

    if (n == 0) {
        // nothing to add
        return;
    }

    DeviceScope scope(device_);

    // To avoid multiple re-allocations, ensure we have enough storage
    // available
    data_->reserve(n, resources_->getDefaultStream(device_));

    // If we're not operating in float16 mode, we don't need the input
    // data to be resident on our device; we can add directly.
    addImpl_(n, x, nullptr);
}

bool
GpuIndexInt8Flat::addImplRequiresIDs_() const {
    return false;
}

void
GpuIndexInt8Flat::addImpl_(int n,
                           const int8_t* x,
                           const Index::idx_t* ids) {
    FAISS_ASSERT(data_);
    FAISS_ASSERT(n > 0);

    // We do not support add_with_ids
    FAISS_THROW_IF_NOT_MSG(!ids, "add_with_ids not supported");

    // Due to GPU indexing in int32, we can't store more than this
    // number of vectors on a GPU
    FAISS_THROW_IF_NOT_FMT(this->ntotal + n <=
                           (faiss::Index::idx_t) std::numeric_limits<int>::max(),
                           "GPU index only supports up to %zu indices",
                           (size_t) std::numeric_limits<int>::max());

    data_->add(x, n, resources_->getDefaultStream(device_));
    this->ntotal += n;
}

void
GpuIndexInt8Flat::searchImpl_(int n,
                              const int8_t* x,
                              int k,
                              int* distances,
                              Index::idx_t* labels) const {
    auto stream = resources_->getDefaultStream(device_);

    // Input and output data are already resident on the GPU
    Tensor<int8_t, 2, true> queries(const_cast<int8_t *>(x), {n, (int) this->d});
    Tensor<int, 2, true> outDistances(distances, {n, k});
    Tensor<Index::idx_t, 2, true> outLabels(labels, {n, k});

    // FlatIndex only supports int indices
    DeviceTensor<int, 2, true> outIntLabels(
            resources_->getMemoryManagerCurrentDevice(), {n, k}, stream);

    data_->query(queries, k, metric_type,
                 outDistances, outIntLabels, true);

    // Convert int to idx_t
    convertTensor<int, faiss::Index::idx_t, 2>(stream,
                                               outIntLabels,
                                               outLabels);
}

void
GpuIndexInt8Flat::reconstruct(faiss::Index::idx_t key,
                              int8_t* out) const {
    DeviceScope scope(device_);

    FAISS_THROW_IF_NOT_MSG(key < this->ntotal, "index out of bounds");
    auto stream = resources_->getDefaultStream(device_);

    auto vec = data_->getVectorsRef()[key];
    fromDevice(vec.data(), out, this->d, stream);
}

void
GpuIndexInt8Flat::reconstruct_n(faiss::Index::idx_t i0,
                                faiss::Index::idx_t num,
                                int8_t* out) const {
    DeviceScope scope(device_);

    FAISS_THROW_IF_NOT_MSG(i0 < this->ntotal, "index out of bounds");
    FAISS_THROW_IF_NOT_MSG(i0 + num - 1 < this->ntotal, "num out of bounds");
    auto stream = resources_->getDefaultStream(device_);

    auto vec = data_->getVectorsRef()[i0];
    fromDevice(vec.data(), out, this->d * num, stream);
}

}}