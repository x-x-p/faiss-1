//
// Created by root on 5/7/20.
//

#include <faiss/gpu/impl/I8FlatIndex.cuh>
#include <faiss/gpu/impl/Distance.cuh>
#include <faiss/gpu/impl/VectorResidual.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/Transpose.cuh>

namespace faiss { namespace gpu {

I8FlatIndex::I8FlatIndex(GpuResources* res,
                         int dim,
                         bool storeTransposed,
                         MemorySpace space) :
        resources_(res),
        dim_(dim),
        storeTransposed_(storeTransposed),
        space_(space),
        num_(0),
        rawData_(space) {}

int I8FlatIndex::getSize() const {
    return vectors_.getSize(0);
}

int I8FlatIndex::getDim() const {
    return vectors_.getSize(1);
}

void
I8FlatIndex::reserve(size_t numVecs, hipStream_t stream) {
    rawData_.reserve(numVecs * dim_, stream);
}


Tensor<int8_t, 2, true>&
I8FlatIndex::getVectorsRef() {
    // Should not call this unless we are in flreturn vectors_;
    return vectors_;
}

void
I8FlatIndex::query(Tensor<int8_t, 2, true>& input,
                 int k,
                 faiss::MetricType metric,
                 Tensor<int, 2, true>& outDistances,
                 Tensor<int, 2, true>& outIndices,
                 bool exactDistance) {
    bfKnnOnDevice(resources_,
                  getCurrentDevice(),
                  resources_->getDefaultStreamCurrentDevice(),
                  storeTransposed_ ? vectorsTransposed_ : vectors_,
                  !storeTransposed_, // is vectors row major?
                  &norms_,
                  input,
                  true, // input is row major
                  k,
                  metric,
                  0,
                  outDistances,
                  outIndices,
                  !exactDistance);
}

void
I8FlatIndex::reconstruct(Tensor<int, 1, true> &listIds,
                         Tensor<int8_t, 2, true> &vecs) {
    runReconstruct(listIds,
                   getVectorsRef(),
                   vecs,
                   resources_->getDefaultStreamCurrentDevice());
}

void
I8FlatIndex::reconstruct(Tensor<int, 2, true> &listIds,
                         Tensor<int8_t, 3, true> &vecs) {
    auto listIds1 = listIds.downcastOuter<1>();
    auto vecs2 = vecs.downcastOuter<2>();

    reconstruct(listIds1, vecs2);
}

void
I8FlatIndex::add(const int8_t* data, int numVecs, hipStream_t stream) {
    if (numVecs == 0) {
        return;
    }

    rawData_.append((char*) data,
                    (size_t) dim_ * numVecs,
                    stream,
                    true /* reserve exactly */);

    num_ += numVecs;

    DeviceTensor<int8_t, 2, true> vectors(
            (int8_t*) rawData_.data(), {(int) num_, dim_}, space_);
    vectors_ = std::move(vectors);

    if (storeTransposed_) {
        vectorsTransposed_ =
                std::move(DeviceTensor<int8_t, 2, true>({dim_, (int) num_}, space_));
        runTransposeAny(vectors_, 0, 1, vectorsTransposed_, stream);
    }
}


// todo: impl bulk remove
void
I8FlatIndex::del(int id, hipStream_t stream) {
    if(id > num_ - 1) {
        return;
    }

    if(id < num_ - 1){
        //不释放以前申请的
        CUDA_VERIFY(hipMemcpy(
                ((char*)rawData_.data()) + id * dim_,
                ((char*)rawData_.data()) + (num_ - 1) * dim_,
                dim_, //In bytes
                hipMemcpyDeviceToDevice
        ));
    }

    num_ -= 1;
    rawData_.resize(num_ * dim_, stream);

    {
        DeviceTensor<int8_t, 2, true> vectors(
                (int8_t*) rawData_.data(), {(int) num_, dim_}, space_);
        vectors_ = std::move(vectors);
    }

    if (storeTransposed_) {
        {
            vectorsTransposed_ =
                    std::move(DeviceTensor<int8_t, 2, true>({dim_, (int) num_}, space_));
            runTransposeAny(vectors_, 0, 1, vectorsTransposed_, stream);
        }
    }
}

void I8FlatIndex::reset() {
    rawData_.clear();
    vectors_ = std::move(DeviceTensor<int8_t , 2, true>());
    vectorsTransposed_ = std::move(DeviceTensor<int8_t, 2, true>());
    norms_ = std::move(DeviceTensor<int, 1, true>());
    num_ = 0;
}


}}