//
// Created by root on 5/8/20.
//

#include <faiss/gpu/utils/blockselect/BlockSelectImpl.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>

namespace faiss { namespace gpu {

// warp Q to thread Q:
// 1, 1
// 32, 2
// 64, 3
// 128, 3
// 256, 4
// 512, 8
// 1024, 8
// 2048, 8

BLOCK_SELECT_DECL(int, true, 1);
BLOCK_SELECT_DECL(int, true, 32);
BLOCK_SELECT_DECL(int, true, 64);
BLOCK_SELECT_DECL(int, true, 128);
BLOCK_SELECT_DECL(int, true, 256);
BLOCK_SELECT_DECL(int, true, 512);
BLOCK_SELECT_DECL(int, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
BLOCK_SELECT_DECL(int, true, 2048);
#endif

BLOCK_SELECT_DECL(int, false, 1);
BLOCK_SELECT_DECL(int, false, 32);
BLOCK_SELECT_DECL(int, false, 64);
BLOCK_SELECT_DECL(int, false, 128);
BLOCK_SELECT_DECL(int, false, 256);
BLOCK_SELECT_DECL(int, false, 512);
BLOCK_SELECT_DECL(int, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
BLOCK_SELECT_DECL(int, false, 2048);
#endif

void runBlockSelect(Tensor<int, 2, true>& in,
                    Tensor<int, 2, true>& outK,
                    Tensor<int, 2, true>& outV,
                    bool dir, int k, hipStream_t stream) {
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BLOCK_SELECT_CALL(int, true, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_CALL(int, true, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_CALL(int, true, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_CALL(int, true, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_CALL(int, true, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_CALL(int, true, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_CALL(int, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_CALL(int, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BLOCK_SELECT_CALL(int, false, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_CALL(int, false, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_CALL(int, false, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_CALL(int, false, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_CALL(int, false, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_CALL(int, false, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_CALL(int, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_CALL(int, false, 2048);
#endif
        }
    }
}

void runBlockSelectPair(Tensor<int, 2, true>& inK,
                        Tensor<int, 2, true>& inV,
                        Tensor<int, 2, true>& outK,
                        Tensor<int, 2, true>& outV,
                        bool dir, int k, hipStream_t stream) {
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(int, true, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(int, true, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(int, true, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(int, true, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(int, true, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(int, true, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(int, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(int, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(int, false, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(int, false, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(int, false, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(int, false, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(int, false, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(int, false, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(int, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(int, false, 2048);
#endif
        }
    }
}

} } // namespace